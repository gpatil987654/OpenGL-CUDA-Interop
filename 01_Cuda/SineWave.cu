#include "hip/hip_runtime.h"
#include "SineWave.cu.h"
#include <hip/hip_runtime.h>

// cuda kernel for sineWave

__global__ void sinWaveKernel(float4 *pPosition, int width, int height, float time)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float u = (float)x / (float)width;
    float v = (float)y / (float)height;

    // NDC
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    float frequency = 4.0f;
    float w = sinf(u * frequency + time) * cosf(v * frequency + time) * 0.5f;

    pPosition[y * width + x] = make_float4(u, w, v, 1.0f);
}

// User Defined Fucntion to call cuda kernel
void launchCUDAKernel(float4 *pos, int width, int height, float time)
{
    dim3 block(8, 8, 1);
    dim3 grid(width / block.x, height / block.y, 1);

    sinWaveKernel<<<grid, block>>>(pos, width, height, time);
}
